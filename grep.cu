
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define MAX_VALUE 5600
#define MAX_STRING_LENGTH 4096
#define CHECK_ERR(x)                                    \
	if (x != hipSuccess) {                               \
		fprintf(stderr,"%s in %s at line %d\n",             \
				hipGetErrorString(err),__FILE__,__LINE__); \
		exit(-1);                                           \
	}                                                     \
	
//	global variables: 
//	d_A: the huge string buffer in GPU
//	d_B: start position of each line in d_A
//	d_C: length of each line in d_A
//	d_D: stores the pattern in GPU 
char *d_A;
int *d_B;
int *d_C;
char *d_D;
hipError_t err;

//kernel function: each thread gets its corresponding line and search the pattern in the naive way.
__global__ void grep (char* d_A, int * d_B, int * d_C, char * d_D, int arrayLength,int patternLength ) {
	//	get the index of the thread.
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	//	only the thread whose index is less than the 4096 does the search
	if (threadIndex < arrayLength) {
		int flag = 1;
		// match algorithm: naive
		// if the length of each substring is less than the size of the pattern, there is certainly no match. Just return
		if(d_C[threadIndex] <patternLength ){
			flag=0;
			return;
		}
		else {
			//	use a for loop to search from every possible position.
			for(int i = d_B[threadIndex]; i < d_B[threadIndex] + d_C[threadIndex] + 1 - patternLength  ; i ++ ){
				flag =1;
				int k = i ;
				for( int j = 0 ; j < patternLength && flag == 1 ; j ++ ) {
					if(d_A[k + j ]!= d_D[j])
						flag = 0;
				}
				// if match ,print and return.
				if(flag==1) {
					printf("%s\n",d_A+d_B[threadIndex]);
					return ;
				}

			}
		}
	}

}


// allocate GPU memory, only 4 blocks of memory in GPU are required.
void allocDeviceMemory(int totalstringSize ){
	
	err = hipMalloc((char **) &d_A, sizeof(char)*MAX_STRING_LENGTH*MAX_VALUE);
	CHECK_ERR(err);

	err = hipMalloc((int **) &d_B, sizeof(int)*MAX_STRING_LENGTH);
	CHECK_ERR(err);

	err = hipMalloc((int **) &d_C, sizeof(int)*MAX_STRING_LENGTH );
	CHECK_ERR(err);
	
	err = hipMalloc((char **) &d_D, sizeof(char)*MAX_STRING_LENGTH );
	CHECK_ERR(err);


}

// copy the string buffer, start position array and string length array to GPU.
void copytoDeviceMemory(char * result, int * start, int * stringSize, char * pattern, int totalstringSize){
	err = hipMemcpy(d_A, result, sizeof(char)*totalstringSize, hipMemcpyHostToDevice);
	CHECK_ERR(err);
	err = hipMemcpy(d_B, start, sizeof(int)*MAX_STRING_LENGTH, hipMemcpyHostToDevice);
	CHECK_ERR(err);
	err = hipMemcpy(d_C, stringSize, sizeof(int)*MAX_STRING_LENGTH, hipMemcpyHostToDevice);
	CHECK_ERR(err);

}

// before terminates the program, free the GPU memory.
void freeDeviceMemory()
{
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);
}



int main(int argc, char *argv[])
{
	int  lineNumber, n;
	FILE *f;
	f = fopen(argv[1], "r");
	if (f == NULL) {
		printf("can't open %s:", argv[1]);
	}
	
	// 	result is the huge string buffer in CPU end
	//	pattern is the string you are looking for.
	//	start array stores the start position of each string in the buffer.
	//	stringSize sotres each string's real length.
	//	totalstringSize records the total lengths of the current 4096 lines of string.
	char * result = (char *)malloc(sizeof(char)*MAX_VALUE * MAX_STRING_LENGTH);
	char * pattern = (char *)malloc(sizeof(char)*MAX_VALUE);
	int start[MAX_STRING_LENGTH];
	int stringSize[MAX_STRING_LENGTH];
	int totalstringSize=0;


	//	line number the line index the current file descriptor is reading.
	lineNumber = 0;
	n=0;

	// initializing...
	for(int i = 0; i < MAX_STRING_LENGTH; i ++ ) {
		start[i] = 0;
		stringSize[i] = 0;
	}
	totalstringSize = 0;
	strcat(pattern,argv[2]);
	
	
	
	//	allocate memory in GPU.
	allocDeviceMemory(totalstringSize);
	//	pattern only needs to be copied to GPU once. Thus do it first.
	err = hipMemcpy(d_D, pattern, sizeof(char)*strlen(pattern), hipMemcpyHostToDevice);
	CHECK_ERR(err);
	
	// 	each while loop ,we read 4096 lines of strings to the buffer string.
	while (fgets(result+start[lineNumber%MAX_STRING_LENGTH], MAX_VALUE, f) != NULL) {

		n = strlen(result+start[lineNumber%MAX_STRING_LENGTH]);

		
		//	save the current string to the huge buffer string
		if (n > 0 && *(result+start[lineNumber%MAX_STRING_LENGTH] + n-1) == '\n'){
			*(result+start[lineNumber%MAX_STRING_LENGTH] + n-1) = '\0';
		}

		//	save the correct start position and string length.
		if(lineNumber%MAX_STRING_LENGTH < MAX_STRING_LENGTH-1) {
			stringSize[lineNumber%MAX_STRING_LENGTH]=n;
			start[lineNumber%MAX_STRING_LENGTH + 1] = start[lineNumber%MAX_STRING_LENGTH] + stringSize[lineNumber%MAX_STRING_LENGTH];
		}
		else {
			stringSize[lineNumber%MAX_STRING_LENGTH]=n;
		}
		totalstringSize += n;
		lineNumber ++;

		//	send the 4096 lines to GPU and do the searching.
		if(lineNumber % MAX_STRING_LENGTH == 0) {
			//	copy first
			copytoDeviceMemory(result,  start, stringSize,  pattern, totalstringSize);
			grep<<<16,256>>>(d_A, d_B, d_C, d_D, MAX_STRING_LENGTH,strlen(pattern));
			//	reset the buffer and other variables.
			memset(result,'\0',sizeof(result));
			totalstringSize = 0;
			start[0]=0;
		}
	}

	//	send the remaining lines to GPU and do the searching.
	if(lineNumber % MAX_STRING_LENGTH !=0) {
		copytoDeviceMemory(result,  start, stringSize,  pattern, totalstringSize);
		//	only copy lineNumber%MAX_STRING_LENGTH strings to GPU.
		grep<<<16,256>>>(d_A, d_B, d_C, d_D,lineNumber % MAX_STRING_LENGTH,strlen(pattern) );
		//	free the memory finally
		freeDeviceMemory();
	}

	return 0;
}
